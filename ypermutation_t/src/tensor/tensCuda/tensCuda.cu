#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <stdexcept>

#include <vector>
#include <algorithm>


//#include "/home/fanasina/progr_/ptens0neD/tensor/tens0neD/tens0neD.h"

//#include "/home/fanasina/progr_/ptens0neD/tensor/tensCuda/tensCuda.h"
#include "tensor/tensCuda/tensCuda.h"




template<typename T>
void cudaTensorProd(Tensor<T>& M, const Tensor<T>& M0, const Tensor<T>& M1) {
    add(M.Dim, M0.Dim, M1.Dim);
    M.initTensor();

    int* d_imM, * d_imM0, * d_imM1;
    hipError_t  errCu = hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&d_imM1, M1.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM1, M1.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(d_imM1, M1.Dim.dim, M1.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM1, M1.Dim.dim, M1.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    T* e, * e0, * e1;
    errCu = hipMalloc((void**)&e, M.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e, M.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&e0, M0.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e0, M0.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&e1, M1.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e1, M1.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(e1, M1.elements, M1.Dim.size * sizeof(T), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(e1, M1.elements, M1.Dim.size * sizeof(T), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    int BLOCKSIZE = 256;//1024;
    int DIMBLOCKS = (M.Dim.size + BLOCKSIZE - 1) / BLOCKSIZE;
    //int DIMBLOCKS = (M.Dim.size) / BLOCKSIZE;

    d_prodTensor<T> << < DIMBLOCKS, BLOCKSIZE >> > (e, d_imM, M.Dim.rank, M.Dim.size, e0, d_imM0, M0.Dim.rank, M0.Dim.size, e1, d_imM1, M1.Dim.rank);

    errCu = hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipFree(e);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(e0);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(e1);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e1) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM0);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM1);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM1) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
}


//template void cudaTensorProd<double>(Tensor<double>& M, const Tensor<double>& M1, const Tensor<double>& M0);
template void cudaTensorProd<float>(Tensor<float>& M, const Tensor<float>& M1, const Tensor<float>& M0);


template<typename T>
void cudaTensorProdEnd(Tensor<T>& M, const Tensor<T>& M0, const Tensor<T>& M1) {
    add(M.Dim, M0.Dim, M1.Dim);
    M.initTensor();

    int* d_imM, * d_imM0, * d_imM1;
    hipError_t errCu = hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&d_imM1, M1.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM1, M1.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(d_imM1, M1.Dim.dim, M1.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM1, M1.Dim.dim, M1.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    T* e, * e0, * e1;
    errCu = hipMalloc((void**)&e, M.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e, M.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&e0, M0.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e0, M0.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&e1, M1.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e1, M1.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(e1, M1.elements, M1.Dim.size * sizeof(T), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(e1, M1.elements, M1.Dim.size * sizeof(T), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    size_t BLOCKSIZE = 1024;
    size_t DIMBLOCKS = (M.Dim.size + BLOCKSIZE - 1) / BLOCKSIZE;

    d_prodTensorEnd<T> << < DIMBLOCKS, BLOCKSIZE >> > (e, d_imM, M.Dim.rank, M.Dim.size, e0, d_imM0, M0.Dim.rank, M0.Dim.size, e1, d_imM1, M1.Dim.rank);

    hipDeviceSynchronize();

    errCu = hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipFree(e);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(e0);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(e1);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e1) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM0);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM1);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM1) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
}


//template void cudaTensorProd<double>(Tensor<double>& M, const Tensor<double>& M1, const Tensor<double>& M0);
template void cudaTensorProdEnd<float>(Tensor<float>& M, const Tensor<float>& M1, const Tensor<float>& M0);


template<typename T>
void cudapermuteTensor(Tensor<T>& M, const Tensor<T>& M0, permutation p) {
    if (p.size == M0.Dim.rank) {
        M.Dim.rank = M0.Dim.rank;
        M.Dim.size = M0.Dim.size;
        M.Dim.initDim();
        M.initTensor();

        p.permute(M.Dim.dim, M0.Dim.dim);


        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);


        int* d_imM, * d_imM0;
        hipError_t errCu = hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int));
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }

        errCu = hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int));
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }

        errCu = hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }

        errCu = hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }


        T* e, * e0;
        errCu = hipMalloc((void**)&e, M.Dim.size * sizeof(T));
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMalloc((void**)&e, M.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }
        errCu = hipMalloc((void**)&e0, M0.Dim.size * sizeof(T));
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMalloc((void**)&e0, M0.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }


        errCu = hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }


        size_t BLOCKSIZE = 256; //1024;//512;
        size_t DIMBLOCKS = (M.Dim.size + BLOCKSIZE - 1) / BLOCKSIZE;
        dim3 blckSZ, gridSZ;
        blckSZ.x = BLOCKSIZE;
        gridSZ.x = DIMBLOCKS;

        int* invP, * d_invP;
        invP = (int*)malloc(M.Dim.rank * sizeof(int));
        inverseArray(invP, p.perm, M.Dim.rank);
        errCu = hipMalloc((void**)&d_invP, M.Dim.rank * sizeof(int));
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMalloc((void**)&d_invP, M.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }

        errCu = hipMemcpy(d_invP, invP, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMemcpy(d_invP, invP, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }
        //printf("size: %ld\n", M.Dim.size);

        //d_prodTensorEnd<T> << < DIMBLOCKS, BLOCKSIZE >> > (e, d_imM, M.Dim.rank, M.Dim.size, e0, d_imM0, M0.Dim.rank, e1, d_imM1, M1.Dim.rank);
        //d_TensorContractnReverseProd<T> << < DIMBLOCKS, BLOCKSIZE >> > (e, d_imM, M.Dim.rank, M.Dim.size, d_imdM, dM.rank, dM.size, e0, d_imM0, M0.Dim.rank, e1, d_imM1, M1.Dim.rank, nestingDepth);
        //d_TensorContractnReverseProd<T> << < gridSZ, blckSZ, 0, 0 >> > (e, d_imM, M.Dim.rank, M.Dim.size, d_imdM, dM.rank, dM.size, e0, d_imM0, M0.Dim.rank, e1, d_imM1, M1.Dim.rank, nestingDepth);
        d_PermLinearTransformCoord<T> << < gridSZ, blckSZ, 0, 0 >> > (e, d_imM, M.Dim.rank, M.Dim.size, e0, d_imM0, M0.Dim.rank, M0.Dim.size, d_invP);
        //d_PermLinearTransformCoord<T> << < gridSZ, blckSZ, 0, 0 >> > (e, d_imM, M.Dim.rank, M.Dim.size, e0, d_imM0, M0.Dim.rank, M0.Dim.size, p.perm);
        //hipDeviceSynchronize();


        errCu = hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }

        errCu = hipFree(e);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipFree(e) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }
        errCu = hipFree(e0);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipFree(e0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }

        errCu = hipFree(d_imM);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipFree(d_imM) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }
        errCu = hipFree(d_imM0);
        if (hipSuccess != errCu) {
            printf("device fnc failed hipFree(d_imM0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
            exit(errCu);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("ellaps time cuda permute tensor: %f ms\n", milliseconds);

    }
}

template
void cudapermuteTensor(Tensor<float>& M, const Tensor<float>& M0, permutation p);


// strict match contract ! if no strict, we take the minimum
template<typename T>
void cudaTensorContractNestProd(Tensor<T>& M, const Tensor<T>& M0, const Tensor<T>& M11, int nestingDepth, bool strict) {


    int perm[M11.Dim.rank];
    struct Tensor<T> M1;
    if (scanPermuteMatchContractTensorfromSrcToDst(perm, M11, M0, nestingDepth)) {
        for (int i = 0; i < M11.Dim.rank; i++) printf(" %d[%d] ", i, perm[i]); printf(": last perm \n");
        struct permutation p(M11.Dim.rank, perm);
        permuteTensor(M1, M11, p);
        M1.Dim.print();

    }
    else {
        printf("Failed in Deep = %d\n", nestingDepth);
        //throw std::check_ProdTensor(" Failed imbrication order in Multiplication matrix ");

        throw std::invalid_argument(" Failed imbrication order in Multiplication matrix ");
        exit(1);
    }


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int len0 = M0.Dim.rank - nestingDepth;
    int len1 = M1.Dim.rank - nestingDepth;

    int* tsub0 = new int[len0];
    int* tsub1 = new int[len1];
    int* tDk1 = new int[nestingDepth];
    int* tDk0 = new int[nestingDepth];
    subArray(tsub0, M0.Dim.dim, 0, len0, 0);
    subArray(tsub1, M1.Dim.dim, 0, len1, nestingDepth);
    subArray(tDk1, M1.Dim.dim, 0, nestingDepth, 0);
    subArray(tDk0, M0.Dim.dim, 0, nestingDepth, len0);

    dimension dSub0(len0, tsub0);
    dimension dSub1(len1, tsub1);
    dimension dM1(nestingDepth, tDk1);
    dimension dM0(nestingDepth, tDk0);
    dimension dM(dM0);
    //bool rev;
    //minReverse(dM, dM0, dM1, rev);
    //if (rev) reverseArray(dM.dim, dM.rank);
    //max(dM, dM0, dM1);

    add(M.Dim, dSub0, dSub1);
    M.initTensor();



    int* d_imM, * d_imM0, * d_imM1, * d_imdM;
    hipError_t errCu = hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM, M.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&d_imdM, dM.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imdM, dM.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM0, M0.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&d_imM1, M1.Dim.rank * sizeof(int));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&d_imM1, M1.Dim.rank * sizeof(int)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM, M.Dim.dim, M.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(d_imdM, dM.dim, dM.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imdM, dM.dim, dM.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM0, M0.Dim.dim, M0.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(d_imM1, M1.Dim.dim, M1.Dim.rank * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(d_imM1, M1.Dim.dim, M1.Dim.rank * sizeof(int), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    T* e, * e0, * e1;
    errCu = hipMalloc((void**)&e, M.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e, M.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&e0, M0.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e0, M0.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMalloc((void**)&e1, M1.Dim.size * sizeof(T));
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMalloc((void**)&e1, M1.Dim.size * sizeof(T)) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(e0, M0.elements, M0.Dim.size * sizeof(T), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipMemcpy(e1, M1.elements, M1.Dim.size * sizeof(T), hipMemcpyHostToDevice);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(e1, M1.elements, M1.Dim.size * sizeof(T), hipMemcpyHostToDevice) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    size_t BLOCKSIZE = 256; //1024;//512;
    size_t DIMBLOCKS = (M.Dim.size + BLOCKSIZE - 1) / BLOCKSIZE;
    dim3 blckSZ, gridSZ;
    blckSZ.x = BLOCKSIZE;
    gridSZ.x = DIMBLOCKS;


    //d_prodTensorEnd<T> << < DIMBLOCKS, BLOCKSIZE >> > (e, d_imM, M.Dim.rank, M.Dim.size, e0, d_imM0, M0.Dim.rank, e1, d_imM1, M1.Dim.rank);
    //d_TensorContractnReverseProd<T> << < DIMBLOCKS, BLOCKSIZE >> > (e, d_imM, M.Dim.rank, M.Dim.size, d_imdM, dM.rank, dM.size, e0, d_imM0, M0.Dim.rank, e1, d_imM1, M1.Dim.rank, nestingDepth);
    //d_TensorContractnReverseProd<T> << < gridSZ, blckSZ, 0, 0 >> > (e, d_imM, M.Dim.rank, M.Dim.size, d_imdM, dM.rank, dM.size, e0, d_imM0, M0.Dim.rank, e1, d_imM1, M1.Dim.rank, nestingDepth);
    d_TensorContractnReverseProd<T> << < gridSZ, blckSZ, 0, 0 >> > (e, d_imM, M.Dim.rank, M.Dim.size, e0, M0.Dim.rank, M0.Dim.size, e1, M1.Dim.rank, M1.Dim.size, d_imdM, dM.rank, dM.size);

    //hipDeviceSynchronize();


    errCu = hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipMemcpy(M.elements, e, M.Dim.size * sizeof(T), hipMemcpyDeviceToHost) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }

    errCu = hipFree(e);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(e0);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(e1);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(e1) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM0);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM0) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    errCu = hipFree(d_imM1);
    if (hipSuccess != errCu) {
        printf("device fnc failed hipFree(d_imM1) \n ErrorCuda: %d : %s\n", errCu, hipGetErrorString(errCu));
        exit(errCu);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ellaps time cuda prod contract prod: %f ms\n", milliseconds);


}

template
void cudaTensorContractNestProd<float>(Tensor<float>& M, const Tensor<float>& M0, const Tensor<float>& M1, int nestingDepth, bool strict);
//template void cudaTensorContractnReverseProd<double>(Tensor<double>& M, const Tensor<double>& M0, const Tensor<double>& M1, int nestingDepth);

