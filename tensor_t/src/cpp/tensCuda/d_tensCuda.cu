#include "hip/hip_runtime.h"
/*#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
*/

#include "d_tensCuda.h"
//#include "index.h"
#include <stdio.h>

////////////////////////////////////////////////////////

//1D grid of 1D blocks
__device__
int d_getGlobalIdx_1D_1D() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}
//1D grid of 2D blocks
__device__
int d_getGlobalIdx_1D_2D() {
    return blockIdx.x * blockDim.x * blockDim.y
        + threadIdx.y * blockDim.x + threadIdx.x;
}
//1D grid of 3D blocks
__device__
int d_getGlobalIdx_1D_3D() {
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z
        + threadIdx.z * blockDim.y * blockDim.x
        + threadIdx.y * blockDim.x + threadIdx.x;
}
//2D grid of 1D blocks
__device__ int d_getGlobalIdx_2D_1D() {
    int blockId
        = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}
//2D grid of 2D blocks
__device__
int d_getGlobalIdx_2D_2D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
//2D grid of 3D blocks
__device__
int d_getGlobalIdx_2D_3D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
//3D grid of 1D blocks
__device__
int d_getGlobalIdx_3D_1D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}
//3D grid of 2D blocks
__device__
int d_getGlobalIdx_3D_2D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
//3D grid of 3D blocks
__device__
int d_getGlobalIdx_3D_3D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}


///////////////////////////////////////////////////////////////////////////


__device__ void d_LinearToCoordEnd(int* ret, size_t lin, int* dim, int rank, size_t size) {
    size_t sm = lin;
    size_t pp = size;
    for (int i = rank - 1;i > 0; --i) {
        pp /= dim[i];
        ret[i] = sm / pp;
        sm %= pp;
    }
    ret[0] = sm;
}

__device__ size_t d_CoordToLinearEnd(int* coo, int* dim, int rank) {
    size_t pp = 1;
    size_t sm = 0;
    for (int i = 0; i < rank; ++i) {
        sm += (coo[i] * pp);
        pp *= dim[i];
    }
    return sm;
}

__device__ size_t d_CoordToLinear(int* coo, int* dim, int rank) {
    size_t pp = 1;
    size_t sm = 0;
    for (int i = rank - 1; i >= 0; --i) {
        sm += (coo[i] * pp);
        pp *= dim[i];
    }
    return sm;
}



__device__ void d_LinearToCoord(int* ret, size_t lin, int* dim, int rank, size_t size) {
    size_t sm = lin;
    size_t pp = size;
    for (int i = 0; i < rank - 1; ++i) {
        pp /= dim[i];
        ret[i] = sm / pp;
        sm %= pp;
    }
    ret[rank - 1] = sm;
}
/*__device__ void d_LinearToSplitSubrankLimSz(size_t& part0, size_t& part1, size_t lin, int* dim, int rank, int rankA, size_t size, size_t sizeA) {
    size_t sm = lin;
    size_t pp = size;
    size_t s = 0;
    size_t p = sizeA;
    int ret;// = new int[rank];
    for (int i = 0; i < rank; ++i) {
        pp /= dim[i];
        ret = sm / pp;
        p /= dim[i];
        s += ret * p;

        sm %= pp;
        if (i == rankA - 1) {
            part0 = s;
            s = 0;
            p = size / sizeA;
        }

    }
    part1 = s;

}*/
__device__ void d_LinearToSplitSubrankLimSz(size_t& part0, size_t& part1, size_t lin, int* dim, int rank, int rankA, size_t size, size_t sizeA) {
    size_t sm = lin;
    size_t pp = size;
    size_t s = 0;
    size_t p = sizeA;
    int ret;// = new int[rank];
    int i;
    for (i = 0; i < rankA; ++i) {
        pp /= dim[i];
        ret = sm / pp;
        p /= dim[i];
        s += ret * p;

        sm %= pp;

    }
    part0 = s;
    s = 0;
    p = size / sizeA;//sizeB
    for (; i < rank; ++i) {
        pp /= dim[i];
        ret = sm / pp;
        p /= dim[i];
        s += ret * p;

        sm %= pp;

    }

    part1 = s;

}
__device__ void d_LinearToSplitSubrankLimSzEnd(size_t& part0, size_t& part1, size_t lin, int* dim, int rank, int rankA, size_t size, size_t sizeA) {
    size_t sm = lin;
    size_t pp = size;
    size_t s = 0;
    size_t p = sizeA;
    int ret;// = new int[rank];
    for (int i = rank - 1; i >= 0; --i) {
        pp /= dim[i];
        ret = sm / pp;
        p /= dim[i];
        s += ret * p;

        sm %= pp;
        if (i == rankA) {
            part1 = s;
            s = 0;
            p = size / sizeA;
        }

    }
    part0 = s;

}


__device__ void d_subArray(int* dst, int* src, int debDst, int finDst, int debSrc) {
    for (int i = debDst; i < finDst; i++) {
        dst[i] = src[i + debSrc];
    }
}

template<typename T>
__global__ void d_prodTensor(T* C, int* dimC, int rankC, size_t size, T* A, int* dimA, int rankA, size_t sizeA, T* B, int* dimB, int rankB) {
    size_t lin0, lin1;

    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        d_LinearToSplitSubrankLimSz(lin0, lin1, i, dimC, rankC, rankA, size, sizeA);

        C[i] = A[lin0] * B[lin1];

    }
}

template __global__  void d_prodTensor<float>(float* C, int* dimC, int rankC, size_t size, float* A, int* dimA, int rankA, size_t sizeA, float* B, int* dimB, int rankB);

template<typename T>
__global__ void d_prodTensorEnd(T* C, int* dimC, int rankC, size_t size, T* A, int* dimA, int rankA, size_t sizeA, T* B, int* dimB, int rankB) {
    size_t lin0, lin1;

    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        d_LinearToSplitSubrankLimSzEnd(lin0, lin1, i, dimC, rankC, rankA, size, sizeA);

        C[i] = A[lin0] * B[lin1];

    }
}

template __global__  void d_prodTensorEnd<float>(float* C, int* dimC, int rankC, size_t size, float* A, int* dimA, int rankA, size_t sizeA, float* B, int* dimB, int rankB);

__device__ void d_minReverse(int* dim, int& rank, const int* dim0, int rank0, const int* dim1, int rank1, bool& rev) {
    if (rank0 > rank1) {
        rank = rank1;
        for (int i = 0; i < rank1; ++i) dim[i] = dim1[i];
        rev = true;
    }
    else if (rank0 < rank1) {
        rank = rank0;
        for (int i = 0; i < rank1; ++i) dim[i] = dim0[i];
        rev = false;
    }
    else {// rank0 == rank1
        rank = rank0;
        for (int i = 0; i < rank0; i++) {
            if (dim[i] > dim1[rank1 - 1 - i]) dim[i] = dim1[rank1 - 1 - i];
            else dim[i] = dim0[i];
        }
        rev = false;
    }
}

__device__ void d_reverseArray(int* arr, int sz) {
    int* tmp;
    //tmp  = (int*)malloc(sz * sizeof(int));

    tmp = new int[sz];
    if (tmp == NULL) {
        size_t limit = 0;
        hipDeviceGetLimit(&limit, hipLimitStackSize);
        printf("hipLimitStackSize: %u | %d (%d) %d | \n", (unsigned)limit, blockIdx.x, blockDim.x, threadIdx.x);
        hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
        printf("hipLimitPrintfFifoSize: %u | %d (%d) %d | \n", (unsigned)limit, blockIdx.x, blockDim.x, threadIdx.x);
        hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
        printf("hipLimitMallocHeapSize: %u | %d (%d) %d | \n", (unsigned)limit, blockIdx.x, blockDim.x, threadIdx.x);

        printf("error Allocation in    tmp  = (int*)malloc(sz * sizeof(int)); |  |   ");
    }int i = 0;
    for (; i < sz / 2; i++) {
        tmp[i] = arr[i];
        arr[i] = arr[sz - 1 - i];
    }
    for (; i < sz; i++) {
        arr[i] = tmp[sz - 1 - i];
    }
    //free(tmp);
    delete[]tmp;
}

__device__ int d_min(int a, int b) {
    if (a < b) return a;
    return b;
}

__device__ void d_concatArray(int* dst, int* src0, int* src1, int debDst, int debSrc0, int finSrc0, int debSrc1, int finSrc1) {
    int i = debDst;
    for (int j = debSrc0; j < finSrc0; j++) {
        dst[i++] = src0[j];
    }
    for (int j = debSrc1; j < finSrc1; j++) {
        dst[i++] = src1[j];
    }
}



__device__ void d_ConcatLinearToSplitSubrankLimSz(size_t& part0, size_t& part1, size_t lin, int* dim, int rank, int rankA, int rankB, size_t size, size_t sizeA, size_t sizeB, int* dM, int dMrank, size_t dMsize, int ind) {
    size_t sm = lin;
    size_t pp = size;
    size_t s = 0;
    size_t p = sizeA;
    //size_t sz_dA = sizeA / dMsize;
    int rankdA = rankA - dMrank;

    int ret;
    int i;
    for (i = 0; i < rankdA; ++i) {
        pp /= dim[i];
        ret = sm / pp;
        p /= dim[i];
        s += ret * p;
        sm %= pp;
    }
    size_t s1 = 0;

    size_t pb = sizeB / dMsize;
    for (; i < rank; ++i) {
        pp /= dim[i];
        ret = sm / pp;
        pb /= dim[i];
        s1 += ret * pb;
        sm %= pp;
    }

    size_t smd = ind;
    size_t ppb = dMsize;
    //size_t pb = size / sz_dA;
    pb = sizeB;
    p = dMsize;
    for (int j = 0;j < dMrank;j++) {
        ppb /= dM[j];
        ret = smd / ppb;
        p /= dM[j];
        s += ret * p;
        pb /= dM[j];
        s1 += ret * pb;
        smd %= ppb;
    }
    //pp = size / sz_dA;
    part0 = s;
    part1 = s1;
}

__device__ void d_SplitLineardToSubrank(size_t& part0, size_t& part1, size_t lin, int* dim, int rank, int rankA, int rankB, size_t size, size_t sizeA, size_t sizeB, int* dM, int dMrank, size_t dMsize) {
    size_t sm = lin;
    size_t pp = size;
    size_t s = 0;
    size_t p = sizeA;
    //size_t sz_dA = sizeA / dMsize;
    int rankdA = rankA - dMrank;

    int ret;
    int i;
    for (i = 0; i < rankdA; ++i) {
        pp /= dim[i];
        ret = sm / pp;
        p /= dim[i];
        s += ret * p;
        sm %= pp;
    }
    size_t s1 = 0;

    size_t pb = sizeB / dMsize;
    for (; i < rank; ++i) {
        pp /= dim[i];
        ret = sm / pp;
        pb /= dim[i];
        s1 += ret * pb;
        sm %= pp;
    }
    part0 = s;
    part1 = s1;
}


__device__ void d_UnionConcatLinearSplitedSubrank(size_t& part0, size_t& part1, size_t p0, size_t p1, size_t size, size_t sizeB, int* dM, int dMrank, size_t dMsize, int ind) {
    size_t s = p0;
    size_t s1 = p1;
    int ret;
    size_t smd = ind;
    size_t ppb = dMsize;
    //size_t pb = size / sz_dA;
    size_t pb = sizeB;
    size_t p = dMsize;
    for (int j = 0;j < dMrank;j++) {
        ppb /= dM[j];
        ret = smd / ppb;
        p /= dM[j];
        s += ret * p;
        pb /= dM[j];
        s1 += ret * pb;
        smd %= ppb;
    }
    //pp = size / sz_dA;
    part0 = s;
    part1 = s1;
}

template<typename T>
__global__ void d_TensorContractnReverseProd(T* C, int* dimC, int rankC, size_t sizeC, T* A, int rankA, size_t sizeA, T* B, int rankB, size_t sizeB, int* dM, int dMrank, size_t dMsize) {

    size_t p0, p1;
    size_t lin0, lin1;


    //size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t i = d_getGlobalIdx_1D_1D();

    if (i < sizeC) {

        d_SplitLineardToSubrank(p0, p1, i, dimC, rankC, rankA, rankB, sizeC, sizeA, sizeB, dM, dMrank, dMsize);

        C[i] = 0;
        for (size_t k = 0; k < dMsize; k++) {

            d_UnionConcatLinearSplitedSubrank(lin0, lin1, p0, p1, sizeC, sizeB, dM, dMrank, dMsize, k);

            //d_ConcatLinearToSplitSubrankLimSz(lin0, lin1, i, dimC, rankC, rankA, rankB, sizeC, sizeA, sizeB, dM, dMrank, dMsize, k);

            C[i] += A[lin0] * B[lin1];
        }
    }

}

template
__global__ void  d_TensorContractnReverseProd<float>(float* C, int* dimC, int rankC, size_t size, float* A, int rankA, size_t sizeA, float* B, int rankB, size_t sizeB, int* dM, int dMrank, size_t dMsize);

__device__ void d_LinearTransformCoord(size_t& dst, size_t src, int* inversePerm, size_t sizeA, int rankDst, int rankSrc, int* dDst, int* dSrc) {
    size_t sm = src;
    size_t pp = sizeA;
    size_t s = 0;
    size_t p = 1;
    int ret;// = new int[rank];
    int i, j;
    for (i = 0; i < rankSrc; ++i) {
        pp /= dSrc[i];
        ret = sm / pp;
        p = 1;
        for (j = inversePerm[i] + 1; j < rankDst;j++) {
            p *= dDst[j];
        }
        s += ret * p;

        sm %= pp;

    }
    dst = s;
    if (s > sizeA) printf("I have a problem in LinearTransformCoord: s:%ld siez:%ld \n", s, sizeA);

}

template<typename T>
__global__ void d_PermLinearTransformCoord(T* C, int* dimC, int rankC, size_t sizeC, T* A, int* dimA, int rankA, size_t sizeA, int* invPerm) {

    //size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t i = d_getGlobalIdx_1D_1D();

    if (i < sizeC) {
        //printf("<i:%*ld ", 3, i);

        size_t img = 0;
        //printf("<i:%*ld, img:%*ld\n", 3, i, 3, img);
        d_LinearTransformCoord(img, i, invPerm, sizeA, rankC, rankA, dimC, dimA);
        //img = d_LinearTransformCoord(i, invPerm, sizeC, dimC, dimA, rankC);

        if (img < sizeC)
            C[img] = A[i];
        else {
            printf("something wrong in device: i:%ld , s:%ld\n", i, img);
        }
    }

}

template
__global__ void  d_PermLinearTransformCoord<float>(float* C, int* dimC, int rankC, size_t size, float* A, int* dimA, int rankA, size_t sizeA, int* invPerm);

